#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "core/pack/Pack.h"
#include "core/pack/GetInds.h"
#include "core/pack/GetDims.h"
#include "core/utils/CudaErrorCheck.cu"
#include "core/utils/CudaSizes.h"
#include "core/utils/TypesUtils.h"

namespace keops {
	

template<typename TYPE, class FUN>
__global__ void GpuConv1DOnDevice_Chunks(FUN fun, int nx, int ny, TYPE **px, TYPE **py, TYPE **pp) 
{

	// get the index of the current thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	// declare shared mem
	extern __shared__ TYPE yj[];
	
	// get templated dimensions :
	typedef typename FUN::DIMSX DIMSX;  // DIMSX is a "vector" of templates giving dimensions of xi variables
	typedef typename FUN::DIMSY DIMSY;  // DIMSY is a "vector" of templates giving dimensions of yj variables
	typedef typename FUN::DIMSP DIMSP;  // DIMSP is a "vector" of templates giving dimensions of parameters variables
	const int DIMY = DIMSY::SUM;        // DIMY  is sum of dimensions for yj variables
	const int DIMP = DIMSP::SUM;        // DIMP  is sum of dimensions for parameters variables
	const int DIMOUT = FUN::DIM; // dimension of output variable
	const int DIMRED = FUN::DIMRED; // dimension of reduction operation
	const int DIMFOUT = DIMSX::FIRST;     // DIMFOUT is dimension of output variable of inner function
	
	static_assert(DIMFOUT == 1, "implemented only for dim 1.");
	
	const int NCHUNKS = 1 + (DIMY-1) / DIMCHUNK;
	const int DIMLASTCHUNK = DIMY - (NCHUNKS-1)*DIMCHUNK;
	
	using F0 = typename FUN::F::template ReplaceVars2 < Var<0,DIMY,0>, Var<0,DIMCHUNK,0>, Var<1,DIMY,1>, Var<1,DIMCHUNK,1> >;
	using FUN_CHUNK = typename F0::template ReplaceVars2 < Var<1,DIMY,0>, Var<1,DIMCHUNK,0>, Var<0,DIMY,1>, Var<0,DIMCHUNK,1> >;
	
	using F1 = typename FUN::F::template ReplaceVars2 < Var<0,DIMY,0>, Var<0,DIMLASTCHUNK,0>, Var<1,DIMY,1>, Var<1,DIMLASTCHUNK,1> >;
	using FUN_LASTCHUNK = typename F1::template ReplaceVars2 < Var<1,DIMY,0>, Var<1,DIMLASTCHUNK,0>, Var<0,DIMY,1>, Var<0,DIMLASTCHUNK,1> >;
	
	// load parameter(s)
	TYPE param_loc[DIMP < 1 ? 1 : DIMP];
	load<DIMSP>(0, param_loc, pp); // load parameters variables from global memory to local thread memory
	
	// get the value of variable (index with i)
	TYPE xi[1+DIMCHUNK];
	TYPE fout[CUDA_BLOCK_SIZE_CHUNKS];
	
	__TYPEACC__ acc[DIMRED];

	if (i < nx) 
	{
		typename FUN::template InitializeReduction<__TYPEACC__, TYPE >()(acc); // acc = 0
	}
	__syncthreads();

	for (int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) 
	{
		// get the current column
		int j = tile * blockDim.x + threadIdx.x;
	
		if (i < nx) 
		{ // we compute x1i only if needed
			for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++) 
			{
				fout[jrel] = 0;
			}
		}
		__syncthreads();
	
		// looping on chunks (except the last)
		#pragma unroll
		for (int chunk=0; chunk<NCHUNKS-1; chunk++) 
		{
			if (i < nx) 
			{
				#pragma unroll
				for (int k=0; k<DIMCHUNK; k++) 
					xi[k+1] = px[1][i*DIMSX::NEXT::FIRST+chunk*DIMCHUNK+k];  // load xi variable from global memory to local thread memory
			}
			__syncthreads();
	
			if (j < ny) 
			{ // we load yj from device global memory only if j<ny
				#pragma unroll
				for (int k=0; k<DIMCHUNK; k++) 
					yj[threadIdx.x * DIMCHUNK+k] = py[0][j*DIMSY::FIRST+chunk*DIMCHUNK+k];
			}
			__syncthreads();
	
			if (i < nx) 
			{ // we compute x1i only if needed
				TYPE * yjrel = yj; // Loop on the columns of the current block.
				for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++, yjrel += DIMCHUNK) 
				{
					call<pack<DIMFOUT,DIMCHUNK>, pack<DIMCHUNK>, DIMSP>(FUN_CHUNK::template EvalFun<FUN::INDS>(),
	                                  xi,
	                                  yjrel,
	                                  param_loc); // Call the function, which outputs results in xi[0:DIMX1]
					fout[jrel] += xi[0];
				}
			}
			__syncthreads();
		}
	
		// last chunk
		{
			if (i < nx) 
			{
				#pragma unroll
				for (int k=0; k<DIMLASTCHUNK; k++) 
					xi[k+1] = px[1][i*DIMSX::NEXT::FIRST+(NCHUNKS-1)*DIMCHUNK+k];  // load xi variable from global memory to local thread memory
			}
			__syncthreads();
	
			if (j < ny) 
			{ // we load yj from device global memory only if j<ny
				#pragma unroll
				for (int k=0; k<DIMLASTCHUNK; k++) 
					yj[threadIdx.x * DIMCHUNK+k] = py[0][j*DIMSY::FIRST+(NCHUNKS-1)*DIMCHUNK+k];
			}
			__syncthreads();
	
			if (i < nx) 
			{ // we compute x1i only if needed
				TYPE * yjrel = yj; // Loop on the columns of the current block.
				for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++, yjrel += DIMCHUNK) 
				{
					call<pack<DIMFOUT,DIMLASTCHUNK>, pack<DIMLASTCHUNK>, DIMSP>(FUN_LASTCHUNK::template EvalFun<FUN::INDS>(),
								  xi,
								  yjrel,
								  param_loc); // Call the function, which outputs results in xi[0:DIMX1]
					fout[jrel] += xi[0];
					typename FUN::template ReducePairShort<__TYPEACC__,TYPE>()(acc, fout+jrel, jrel + tile * blockDim.x);     // acc += xi
				}
			}
			__syncthreads();
		}

	}

	if (i < nx) {
		typename FUN::template FinalizeOutput<__TYPEACC__,TYPE>()(acc, px[0] + i * DIMOUT, px, i);
	}
	__syncthreads();
}


template<typename TYPE, class FUN>
__global__ void GpuConv1DOnDevice(FUN fun, int nx, int ny, TYPE **px, TYPE **py, TYPE **pp) {

  // get the index of the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // declare shared mem
  extern __shared__ TYPE yj[];

  // get templated dimensions :
  typedef typename FUN::DIMSX DIMSX;  // DIMSX is a "vector" of templates giving dimensions of xi variables
  typedef typename FUN::DIMSY DIMSY;  // DIMSY is a "vector" of templates giving dimensions of yj variables
  typedef typename FUN::DIMSP DIMSP;  // DIMSP is a "vector" of templates giving dimensions of parameters variables
  const int DIMX = DIMSX::SUM;        // DIMX  is sum of dimensions for xi variables
  const int DIMY = DIMSY::SUM;        // DIMY  is sum of dimensions for yj variables
  const int DIMP = DIMSP::SUM;        // DIMP  is sum of dimensions for parameters variables
  const int DIMOUT = FUN::DIM; // dimension of output variable
  const int DIMRED = FUN::DIMRED; // dimension of reduction operation
  const int DIMFOUT = DIMSX::FIRST;     // DIMFOUT is dimension of output variable of inner function

  // load parameter(s)
  TYPE param_loc[DIMP < 1 ? 1 : DIMP];
  load<DIMSP>(0, param_loc, pp); // load parameters variables from global memory to local thread memory

  // get the value of variable (index with i)
  TYPE xi[DIMX < 1 ? 1 : DIMX];
  __TYPEACC__ acc[DIMRED];
#if SUM_SCHEME == BLOCK_SUM
    // additional tmp vector to store intermediate results from each block
    TYPE tmp[DIMRED];
#elif SUM_SCHEME == KAHAN_SCHEME
    // additional tmp vector to accumulate errors
    const int DIM_KAHAN = FUN::template KahanScheme<__TYPEACC__,TYPE>::DIMACC;
    TYPE tmp[DIM_KAHAN];
#endif
  if (i < nx) {
    typename FUN::template InitializeReduction<__TYPEACC__, TYPE >()(acc); // acc = 0
#if SUM_SCHEME == KAHAN_SCHEME
    VectAssign<DIM_KAHAN>(tmp,0.0f);
#endif
    load<typename DIMSX::NEXT>(i, xi + DIMFOUT, px + 1); // load xi variables from global memory to local thread memory
  }

  for (int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) {

    // get the current column
    int j = tile * blockDim.x + threadIdx.x;

    if (j < ny) { // we load yj from device global memory only if j<ny
      load<DIMSY>(j, yj + threadIdx.x * DIMY, py); // load yj variables from global memory to shared memory
    }
    __syncthreads();

    if (i < nx) { // we compute x1i only if needed
      TYPE * yjrel = yj; // Loop on the columns of the current block.
#if SUM_SCHEME == BLOCK_SUM
      typename FUN::template InitializeReduction<TYPE,TYPE>()(tmp); // tmp = 0
#endif
      for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++, yjrel += DIMY) {
        call<DIMSX, DIMSY, DIMSP>(fun,
                                  xi,
                                  yjrel,
                                  param_loc); // Call the function, which outputs results in xi[0:DIMX1]
#if SUM_SCHEME == BLOCK_SUM
#if USE_HALF
        int ind = jrel + tile * blockDim.x;
        typename FUN::template ReducePairShort<TYPE,TYPE>()(tmp, xi, __floats2half2_rn(2*ind,2*ind+1));     // tmp += xi
#else
        typename FUN::template ReducePairShort<TYPE,TYPE>()(tmp, xi, jrel + tile * blockDim.x);     // tmp += xi
#endif
#elif SUM_SCHEME == KAHAN_SCHEME
        typename FUN::template KahanScheme<__TYPEACC__,TYPE>()(acc, xi, tmp);     
#else
#if USE_HALF
        int ind = jrel + tile * blockDim.x;
        typename FUN::template ReducePairShort<__TYPEACC__,TYPE>()(acc, xi, __floats2half2_rn(2*ind,2*ind+1));     // acc += xi
#else
	typename FUN::template ReducePairShort<__TYPEACC__,TYPE>()(acc, xi, jrel + tile * blockDim.x);     // acc += xi
#endif
#endif
      }
#if SUM_SCHEME == BLOCK_SUM
      typename FUN::template ReducePair<__TYPEACC__,TYPE>()(acc, tmp);     // acc += tmp
#endif
    }
    __syncthreads();
  }
  if (i < nx) {
    typename FUN::template FinalizeOutput<__TYPEACC__,TYPE>()(acc, px[0] + i * DIMOUT, px, i);
  }

}


struct GpuConv1D_FromHost {

  template<typename TYPE, class FUN>
  static int Eval_(FUN fun, int nx, int ny, TYPE **px_h, TYPE **py_h, TYPE **pp_h) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;
    const int DIMP = DIMSP::SUM;
    const int DIMOUT = FUN::DIM; // dimension of output variable
    const int DIMFOUT = DIMSX::FIRST;     // DIMFOUT is dimension of output variable of inner function
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // pointers to device data
    TYPE *x_d, *y_d, *param_d;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    CudaSafeCall(hipMalloc((void **) &p_data,
                            sizeof(TYPE *) * (SIZEI + SIZEJ + SIZEP)
                                + sizeof(TYPE) * (DIMP + nx * (DIMX - DIMFOUT + DIMOUT) + ny * DIMY)));

    TYPE **p_data_a = (TYPE **) p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;
    p_data_a += SIZEP;
    TYPE *p_data_b = (TYPE *) p_data_a;
    param_d = p_data_b;
    p_data_b += DIMP;
    x_d = p_data_b;
    p_data_b += nx * (DIMX - DIMFOUT + DIMOUT);
    y_d = p_data_b;

    // host arrays of pointers to device data
    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    int nvals;
    // if DIMSP is empty (i.e. no parameter), nvals = -1 which could result in a segfault
    if (SIZEP > 0) {
      php_d[0] = param_d;
      nvals = DIMSP::VAL(0);
      CudaSafeCall(hipMemcpy(php_d[0], pp_h[0], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));

      for (int k = 1; k < SIZEP; k++) {
        php_d[k] = php_d[k - 1] + nvals;
        nvals = DIMSP::VAL(k);
        CudaSafeCall(hipMemcpy(php_d[k], pp_h[k], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));
      }
    }

    phx_d[0] = x_d;
    nvals = nx * DIMOUT;
    for (int k = 1; k < SIZEI; k++) {
      phx_d[k] = phx_d[k - 1] + nvals;
      nvals = nx * DIMSX::VAL(k);
      CudaSafeCall(hipMemcpy(phx_d[k], px_h[k], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));
    }

    // if DIMSY is empty (i.e. no Vj variable), nvals = -1 which could result in a segfault
    if (SIZEJ > 0) {
      phy_d[0] = y_d;
      nvals = ny * DIMSY::VAL(0);
      CudaSafeCall(hipMemcpy(phy_d[0], py_h[0], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));

      for (int k = 1; k < SIZEJ; k++) {
        phy_d[k] = phy_d[k - 1] + nvals;
        nvals = ny * (int) DIMSY::VAL(k);
        CudaSafeCall(hipMemcpy(phy_d[k], py_h[k], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));
      }
    }

    // copy arrays of pointers
    CudaSafeCall(hipMemcpy(pp_d, php_d, SIZEP * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(px_d, phx_d, SIZEI * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(py_d, phy_d, SIZEJ * sizeof(TYPE *), hipMemcpyHostToDevice));

    // Compute on device : grid and block are both 1d
    int dev = -1;
    CudaSafeCall(hipGetDevice(&dev));

    dim3 blockSize;

    SetGpuProps(dev);

#if ENABLECHUNK // register pressure case...
      blockSize.x = CUDA_BLOCK_SIZE_CHUNKS;
#else
	  // warning : blockSize.x was previously set to CUDA_BLOCK_SIZE; currently CUDA_BLOCK_SIZE value is used as a bound.
      blockSize.x = ::std::min(CUDA_BLOCK_SIZE,
                             ::std::min(maxThreadsPerBlock,
                                        (int) (sharedMemPerBlock / ::std::max(1,
                                                                              (int) (  DIMY
                                                                                  * sizeof(TYPE)))))); // number of threads in each block
#endif
    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

#if ENABLECHUNK
      GpuConv1DOnDevice_Chunks<TYPE> 
		  <<< gridSize, blockSize, blockSize.x * DIMCHUNK * sizeof(TYPE) >>> 
			  (fun, nx, ny, px_d, py_d, pp_d);
#else
      GpuConv1DOnDevice<TYPE> 
		  <<< gridSize, blockSize, blockSize.x * DIMY * sizeof(TYPE) >>> 
			  (fun, nx, ny, px_d, py_d, pp_d);
#endif

    // block until the device has completed
    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();

    // Send data from device to host.
    CudaSafeCall(hipMemcpy(*px_h, x_d, sizeof(TYPE) * (nx * DIMOUT), hipMemcpyDeviceToHost));

    // Free memory.
    CudaSafeCall(hipFree(p_data));

    return 0;
  }

// and use getlist to enroll them into "pointers arrays" px and py.
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, int device_id, TYPE *x1_h, Args... args) {

    if (device_id != -1)
      CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    getlist<INDSI>(px_h + 1, args...);
    getlist<INDSJ>(py_h, args...);
    getlist<INDSP>(pp_h, args...);

    return Eval_(fun, nx, ny, px_h, py_h, pp_h);

  }

// same without the device_id argument
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_h, Args... args) {
    return Eval(fun, nx, ny, -1, x1_h, args...);
  }

// Idem, but with args given as an array of arrays, instead of an explicit list of arrays
  template<typename TYPE, class FUN>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_h, TYPE **args, int device_id = -1) {

    // We set the GPU device on which computations will be performed
    if (device_id != -1)
      CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    for (int i = 1; i < SIZEI; i++)
      px_h[i] = args[INDSI::VAL(i - 1)];
    for (int i = 0; i < SIZEJ; i++)
      py_h[i] = args[INDSJ::VAL(i)];
    for (int i = 0; i < SIZEP; i++)
      pp_h[i] = args[INDSP::VAL(i)];

    return Eval_(fun, nx, ny, px_h, py_h, pp_h);

  }

};

struct GpuConv1D_FromDevice {
  template<typename TYPE, class FUN>
  static int Eval_(FUN fun, int nx, int ny, TYPE **phx_d, TYPE **phy_d, TYPE **php_d) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    CudaSafeCall(hipMalloc((void **) &p_data, sizeof(TYPE *) * (SIZEI + SIZEJ + SIZEP)));

    TYPE **p_data_a = (TYPE **) p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;

    CudaSafeCall(hipMemcpy(px_d, phx_d, SIZEI * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(py_d, phy_d, SIZEJ * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(pp_d, php_d, SIZEP * sizeof(TYPE *), hipMemcpyHostToDevice));

    // Compute on device : grid and block are both 1d

    int dev = -1;
    CudaSafeCall(hipGetDevice(&dev));

    SetGpuProps(dev);

    dim3 blockSize;
#if ENABLECHUNK  // register pressure case...
      blockSize.x = CUDA_BLOCK_SIZE_CHUNKS;
#else
      const int DIMY = DIMSY::SUM;
	  // warning : blockSize.x was previously set to CUDA_BLOCK_SIZE; currently CUDA_BLOCK_SIZE value is used as a bound.
      blockSize.x = ::std::min(CUDA_BLOCK_SIZE,
                             ::std::min(maxThreadsPerBlock,
                                        (int) (sharedMemPerBlock / ::std::max(1,
                                                                              (int) (  DIMY
                                                                                  * sizeof(TYPE)))))); // number of threads in each block
#endif
	
    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

#if ENABLECHUNK
      GpuConv1DOnDevice_Chunks<TYPE> 
		  <<< gridSize, blockSize, blockSize.x * DIMCHUNK * sizeof(TYPE) >>> 
			  (fun, nx, ny, px_d, py_d, pp_d);
#else
      GpuConv1DOnDevice<TYPE> <<< gridSize, blockSize, blockSize.x * DIMY * sizeof(TYPE) >>> 
		  (fun, nx, ny, px_d, py_d, pp_d);
#endif

    // block until the device has completed
    CudaSafeCall(hipDeviceSynchronize());

    CudaCheckError();

    CudaSafeCall(hipFree(p_data));

    return 0;
  }

// Same wrappers, but for data located on the device
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, int device_id, TYPE *x1_d, Args... args) {

    // device_id is provided, so we set the GPU device accordingly
    // Warning : is has to be consistent with location of data
    CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    phx_d[0] = x1_d;

    getlist<INDSI>(phx_d + 1, args...);
    getlist<INDSJ>(phy_d, args...);
    getlist<INDSP>(php_d, args...);

    return Eval_(fun, nx, ny, phx_d, phy_d, php_d);

  }

// same without the device_id argument
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_d, Args... args) {
    // We set the GPU device on which computations will be performed
    // to be the GPU on which data is located.
    // NB. we only check location of x1_d which is the output vector
    // so we assume that input data is on the same GPU
    // note : hipPointerGetAttributes has a strange behaviour:
    // it looks like it makes a copy of the vector on the default GPU device (0) !!! 
    // So we prefer to avoid this and provide directly the device_id as input (first function above)
    hipPointerAttribute_t attributes;
    CudaSafeCall(hipPointerGetAttributes(&attributes, x1_d));
    return Eval(fun, nx, ny, attributes.device, x1_d, args...);
  }

  template<typename TYPE, class FUN>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_d, TYPE **args, int device_id = -1) {

    if (device_id == -1) {
      // We set the GPU device on which computations will be performed
      // to be the GPU on which data is located.
      // NB. we only check location of x1_d which is the output vector
      // so we assume that input data is on the same GPU
      // note : hipPointerGetAttributes has a strange behaviour:
      // it looks like it makes a copy of the vector on the default GPU device (0) !!!
      // So we prefer to avoid this and provide directly the device_id as input (else statement below)
      hipPointerAttribute_t attributes;
      CudaSafeCall(hipPointerGetAttributes(&attributes, x1_d));
      CudaSafeCall(hipSetDevice(attributes.device));
    } else // device_id is provided, so we use it. Warning : is has to be consistent with location of data
      CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_d[SIZEI];
    TYPE *py_d[SIZEJ];
    TYPE *pp_d[SIZEP];

    px_d[0] = x1_d;
    for (int i = 1; i < SIZEI; i++)
      px_d[i] = args[INDSI::VAL(i - 1)];
    for (int i = 0; i < SIZEJ; i++)
      py_d[i] = args[INDSJ::VAL(i)];
    for (int i = 0; i < SIZEP; i++)
      pp_d[i] = args[INDSP::VAL(i)];

    return Eval_(fun, nx, ny, px_d, py_d, pp_d);

  }

};

}
